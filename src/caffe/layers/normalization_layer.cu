#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void LRNFillScale(const int nthreads, const Dtype* in,
    const int num, const int channels, const int height,
    const int width, const int size, const Dtype alpha_over_size,
    Dtype* scale) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int n = index / width / height;
    int offset = (n * channels * height + h) * width + w;
    int step = height * width;
    in += offset;
    scale += offset;
    int head = 0;
    int pre_pad = (size - 1) / 2;
    int post_pad = size - pre_pad - 1;
    Dtype accum_scale = 0;
    // fill the scale at [n, :, h, w]
    // accumulate values
    while (head < post_pad) {
      accum_scale += in[head * step] * in[head * step];
      ++head;
    }
    // until we reach size, nothing needs to be subtracted
    while (head < size) {
      accum_scale += in[head * step] * in[head * step];
      scale[(head - post_pad) * step] = 1. + accum_scale * alpha_over_size;
      ++head;
    }
    // both add and subtract
    while (head < channels) {
      accum_scale += in[head * step] * in[head * step];
      accum_scale -= in[(head - size) * step] * in[(head - size) * step];
      scale[(head - post_pad) * step] = 1. + accum_scale * alpha_over_size;
      ++head;
    }
    // subtract only
    while (head < channels + post_pad) {
      accum_scale -= in[(head - size) * step] * in[(head - size) * step];
      scale[(head - post_pad) * step] = 1. + accum_scale * alpha_over_size;
      ++head;
    }
  }
}


template <typename Dtype>
void NormalizationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
   int num = bottom[0]->num();
   const Dtype* bottom_data = bottom[0]->gpu_data();
   CHECK(squared_length_.count());
   Dtype* squared_length_data = squared_length_.mutable_gpu_data();
   int count = bottom[0]->count() / num;

   // compute the square of the length
   for (int n = 0; n < num; n++) {
        const Dtype* curr_bottom_data = bottom_data + bottom[0]->offset(n);
        Dtype* curr_squared_length_data = squared_length_data + n;
        caffe_gpu_dot(count, curr_bottom_data, curr_bottom_data, curr_squared_length_data);
   }
   //caffe_gpu_powx(
}


template <typename Dtype>
void NormalizationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
}

INSTANTIATE_CLASS(NormalizationLayer);

}  // namespace caffe
